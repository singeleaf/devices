#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "test_utils.h"
#include <unistd.h>


__global__ void add(float* a, float* b, float* c) {
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}


int main() {
    float *a, *b, *c;
    CHECK_RUNTIME_API(hipMalloc(&a, 1024 * sizeof(float)));
    CHECK_RUNTIME_API(hipMalloc(&b, 1024 * sizeof(float)));
    CHECK_RUNTIME_API(hipMalloc(&c, 1024 * sizeof(float)));

    add<<<1, 1024>>>(a, b, c);
    return 0;
}
